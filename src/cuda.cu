#include "hip/hip_runtime.h"
#include "cuda.cuh"

#include <cstring>

#include "helper.h"

#include ""


///
/// Algorithm storage
///
// Host copy of input image
Image cuda_input_image;
// Host copy of output image
Image cuda_output_image;
// Host copy of image tiles in each dimension
unsigned int cuda_TILES_X, cuda_TILES_Y;
// image data size
size_t image_data_size;

// CUDA device
// Pointer to device buffer for calculating the sum of each tile mosaic, this must be passed to a kernel to be used on device
__device__ unsigned long long* d_mosaic_sum;
// Pointer to device buffer for storing the output pixels of each tile, this must be passed to a kernel to be used on device
__device__ unsigned char* d_mosaic_value;
// Pointer to device image data buffer, for storing the input image, this must be passed to a kernel to be used on device
__device__ unsigned char* d_input_image_data;
// Pointer to device image data buffer, for storing the output image data, this must be passed to a kernel to be used on device
__device__ unsigned char* d_output_image_data;
// Pointer to device buffer for the global pixel average sum, this must be passed to a kernel to be used on device
__device__ unsigned long long* d_global_pixel_sum;

// CUDA constant
__constant__ unsigned int d_CHANNELS;
__constant__ unsigned int d_TILES_X, d_TILES_Y;

unsigned long long* host_mosaic_sum;
unsigned char* host_mosaic_value;

__device__ void array_add_up(int* arr, int* arr2, int* result, unsigned int length) {
	for (unsigned int i = 0; i < length; i++)
		result[i] = arr[i] + arr2[i];
}

__global__ void sum_tile(unsigned char const* __restrict__ input_image_data, unsigned long long* mosaic_sum) {

	/*
	* gridDim.x * blockIdx.y: block number in previous lines
	*/
	const unsigned int tile_index = (gridDim.x * blockIdx.y + blockIdx.x) * d_CHANNELS;
	const unsigned int tile_offset = (TILE_SIZE * TILE_SIZE * gridDim.x * blockIdx.y + TILE_SIZE * blockIdx.x) * d_CHANNELS;
	const unsigned int pixel_offset = (blockDim.x * gridDim.x * threadIdx.y + threadIdx.x) * d_CHANNELS;

	for (unsigned int ch = 0; ch < d_CHANNELS; ch++) {
		const unsigned char pixel = input_image_data[tile_offset + pixel_offset + ch];
		// sum up all the r/g/b channel in current tile
		atomicAdd(&mosaic_sum[tile_index + ch], pixel);
	}
}

__global__ void sum_tile_shfl(unsigned char const* __restrict__ input_image_data, unsigned long long* mosaic_sum) {
	/*
	* gridDim.x * blockIdx.y: block number in previous lines
	*/
	const unsigned int tile_index = (gridDim.x * blockIdx.y + blockIdx.x) * d_CHANNELS;
	const unsigned int tile_offset = (TILE_SIZE * TILE_SIZE * gridDim.x * blockIdx.y + TILE_SIZE * blockIdx.x) * d_CHANNELS;
	const unsigned int pixel_offset = (blockDim.x * gridDim.x * threadIdx.y + threadIdx.x) * d_CHANNELS;

	unsigned long long pixel_r = (unsigned long long)input_image_data[tile_offset + pixel_offset];
	unsigned long long pixel_g = (unsigned long long)input_image_data[tile_offset + pixel_offset + 1];
	unsigned long long pixel_b = (unsigned long long)input_image_data[tile_offset + pixel_offset + 2];

	__syncthreads();

	for (unsigned int offset = 16; offset > 0; offset >>= 1) {
		pixel_r += __shfl_down(pixel_r, offset);
		pixel_g += __shfl_down(pixel_g, offset);
		pixel_b += __shfl_down(pixel_b, offset);
	}

	if (threadIdx.x % 32 == 0) {

		atomicAdd(&mosaic_sum[tile_index], pixel_r);
		atomicAdd(&mosaic_sum[tile_index + 1], pixel_g);
		atomicAdd(&mosaic_sum[tile_index + 2], pixel_b);
	}
}

__global__ void compact_mosaic(unsigned long long* mosaic_sum, unsigned char* mosaic_value, unsigned long long* global_pixel_sum) {
	/*
	* blockIdx.y * gridDim.x * blockDim.x * blockDim.y: Calculate excuted threads based on block y index (multiply whole block size in a grid line)
	* blockIdx.x * blockDim.x * blockDim.y: Calculate fully excuted blocked in current running grid line
	* threadIdx.y * blockDim.x + threadIdx.x: excuted threads in a block
	*/
	unsigned int i = blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	unsigned int mosaic_sum_index = i * d_CHANNELS + threadIdx.z;

	mosaic_value[mosaic_sum_index] = (unsigned char)(mosaic_sum[mosaic_sum_index] / TILE_PIXELS);

	atomicAdd(&global_pixel_sum[threadIdx.z], mosaic_value[mosaic_sum_index]);
}

__global__ void broadcast(unsigned char const* __restrict__ mosaic_value, unsigned char* output_image_data) {
	const unsigned int tile_index = (gridDim.x * blockIdx.y + blockIdx.x) * d_CHANNELS;
	const unsigned int tile_offset = (TILE_SIZE * TILE_SIZE * gridDim.x * blockIdx.y + TILE_SIZE * blockIdx.x) * d_CHANNELS;
	const unsigned int pixel_offset = (blockDim.x * gridDim.x * threadIdx.y + threadIdx.x) * d_CHANNELS;

	for (unsigned int ch = 0; ch < d_CHANNELS; ch++) {
		*(output_image_data + tile_offset + pixel_offset + ch) = mosaic_value[tile_index + ch];
	}
}

void cuda_begin(const Image* input_image) {
	// These are suggested CUDA memory allocations that match the CPU implementation
	// If you would prefer, you can rewrite this function (and cuda_end()) to suit your preference

	cuda_TILES_X = input_image->width / TILE_SIZE;
	cuda_TILES_Y = input_image->height / TILE_SIZE;

	// Allocate buffer for calculating the sum of each tile mosaic
	CUDA_CALL(hipMalloc(&d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned long long)));

	// Allocate buffer for storing the output pixel value of each tile
	CUDA_CALL(hipMalloc(&d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned char)));

	image_data_size = input_image->width * input_image->height * input_image->channels * sizeof(unsigned char);

	// Allocate copy of input image
	cuda_input_image = *input_image;
	cuda_input_image.data = (unsigned char*)malloc(image_data_size);
	memcpy(cuda_input_image.data, input_image->data, image_data_size);

	// Allocate copy of input image
	cuda_output_image = *input_image;
	cuda_output_image.data = (unsigned char*)malloc(image_data_size);

	// Allocate and fill device buffer for storing input image data
	CUDA_CALL(hipMalloc(&d_input_image_data, image_data_size));
	CUDA_CALL(hipMemcpy(d_input_image_data, input_image->data, image_data_size, hipMemcpyHostToDevice));

	// Allocate and fill device buffer for storing input image channels number
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CHANNELS), &input_image->channels, sizeof(int)));
	// Allocate and fill device buffer for storing tile x and y
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_X), &cuda_TILES_X, sizeof(unsigned int)));
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_TILES_Y), &cuda_TILES_Y, sizeof(unsigned int)));

	// Allocate device buffer for storing output image data
	CUDA_CALL(hipMalloc(&d_output_image_data, image_data_size));

	// Allocate and zero buffer for calculation global pixel average
	CUDA_CALL(hipMalloc(&d_global_pixel_sum, input_image->channels * sizeof(unsigned long long)));


	// allocate for skip dunctions
	// Allocate buffer for calculating the sum of each tile mosaic
	host_mosaic_sum = (unsigned long long*)malloc(cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned long long));
	// Allocate buffer for storing the output pixel value of each tile
	host_mosaic_value = (unsigned char*)malloc(cuda_TILES_X * cuda_TILES_Y * input_image->channels * sizeof(unsigned char));
}

void cuda_stage1() {
	// Optionally during development call the skip function with the correct inputs to skip this stage
	// skip_tile_sum(&host_input_image, host_mosaic_sum);

	// init params for kernel
	dim3 blocksPerGrid(cuda_TILES_X, cuda_TILES_Y, 1);
	const unsigned int block_width = (unsigned int)TILE_SIZE;
	// block per grid is equal to the tile_x and tile_y
	dim3 threadsPerBlock(block_width, block_width, 1);

	// init sum array by 0
	CUDA_CALL(hipMemset(d_mosaic_sum, 0, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long)));

	// Run CUDA
	//sum_tile << < blocksPerGrid, threadsPerBlock >> > (d_input_image_data, d_mosaic_sum);
	sum_tile_shfl << < blocksPerGrid, threadsPerBlock >> > (d_input_image_data, d_mosaic_sum);
	hipDeviceSynchronize();

#ifdef VALIDATION
	// TODO: Uncomment and call the validation function with the correct inputs
	// You will need to copy the data back to host before passing to these functions
	// (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
	memset(host_mosaic_sum, 0, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long));
	CUDA_CALL(hipMemcpy(host_mosaic_sum, d_mosaic_sum, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned long long), hipMemcpyDeviceToHost));
	validate_tile_sum(&cuda_input_image, host_mosaic_sum);
#endif
}

void cuda_stage2(unsigned char* output_global_average) {
	// Optionally during development call the skip function with the correct inputs to skip this stage
	// skip_compact_mosaic(cuda_TILES_X, cuda_TILES_Y, host_mosaic_sum, host_mosaic_value, output_global_average);

	unsigned long long* whole_image_sum = (unsigned long long*)malloc(cuda_input_image.channels * sizeof(unsigned long long));
	memset(whole_image_sum, 0, cuda_input_image.channels * sizeof(unsigned long long));

	// blockIdx.z refer to the max channels supported
	dim3 threadsPerBlock(4, 4, 3);
	dim3 blocksPerGrid(cuda_TILES_X / 4, cuda_TILES_Y / 4, 1);

	compact_mosaic << <blocksPerGrid, threadsPerBlock >> > (d_mosaic_sum, d_mosaic_value, d_global_pixel_sum);
	hipDeviceSynchronize();

	CUDA_CALL(hipMemcpy(whole_image_sum, d_global_pixel_sum, cuda_input_image.channels * sizeof(unsigned long long), hipMemcpyDeviceToHost));

	for (unsigned int ch = 0; ch < cuda_input_image.channels; ++ch) {
		output_global_average[ch] = (unsigned char)(whole_image_sum[ch] / (cuda_TILES_X * cuda_TILES_Y));
	}


#ifdef VALIDATION
	// TODO: Uncomment and call the validation functions with the correct inputs
	// You will need to copy the data back to host before passing to these functions
	// (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
	CUDA_CALL(hipMemcpy(host_mosaic_value, d_mosaic_value, cuda_TILES_X * cuda_TILES_Y * cuda_input_image.channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
	validate_compact_mosaic(cuda_TILES_X, cuda_TILES_Y, host_mosaic_sum, host_mosaic_value, output_global_average);
#endif    
}
void cuda_stage3() {
	// Optionally during development call the skip function with the correct inputs to skip this stage
	// skip_broadcast(&cuda_input_image, host_mosaic_value, &host_output_image);

	// init params for kernel
	const unsigned int block_width = (unsigned int)TILE_SIZE;
	// block per grid is equal to the tile_x and tile_y
	dim3 blocksPerGrid(cuda_TILES_X, cuda_TILES_Y, 1);
	dim3 threadsPerBlock(block_width, block_width, 1);

	broadcast << <blocksPerGrid, threadsPerBlock >> > (d_mosaic_value, d_output_image_data);
	hipDeviceSynchronize();

#ifdef VALIDATION
	// TODO: Uncomment and call the validation function with the correct inputs
	// You will need to copy the data back to host before passing to these functions
	// (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
	CUDA_CALL(hipMemcpy(cuda_output_image.data, d_output_image_data, image_data_size, hipMemcpyDeviceToHost));
	validate_broadcast(&cuda_input_image, host_mosaic_value, &cuda_output_image);
#endif    
}
void cuda_end(Image* output_image) {
	// This function matches the provided cuda_begin(), you may change it if desired

	// Store return value
	output_image->width = cuda_input_image.width;
	output_image->height = cuda_input_image.height;
	output_image->channels = cuda_input_image.channels;
	CUDA_CALL(hipMemcpy(output_image->data, d_output_image_data, output_image->width * output_image->height * output_image->channels * sizeof(unsigned char), hipMemcpyDeviceToHost));

	// Release allocations
	free(cuda_input_image.data);
	CUDA_CALL(hipFree(d_mosaic_value));
	CUDA_CALL(hipFree(d_mosaic_sum));
	CUDA_CALL(hipFree(d_global_pixel_sum));
	CUDA_CALL(hipFree(d_input_image_data));
	CUDA_CALL(hipFree(d_output_image_data));
}
